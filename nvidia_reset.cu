// For the CUDA runtime routines (prefixed with "cuda_")
#include <stdio.h>
#include <hip/hip_runtime.h>

/**
 * Reset the NVIDIA device
 *
 * cudaDeviceReset causes the driver to clean up all state. While not mandatory in normal operation, it is good practice.
 * It is also needed to ensure correct operation when the application is being profiled. Calling cudaDeviceReset causes
 * all profile data to be flushed before the application exits
 */
int main(void)
{
    hipError_t err = hipDeviceReset();	// Error code to check return values for CUDA calls

    if (err != hipSuccess) {
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    return 0;
}
